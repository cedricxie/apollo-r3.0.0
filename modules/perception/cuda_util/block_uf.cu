#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2014, Victor Matheus de Araujo Oliveira All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 * Redistributions of source code must retain the above copyright notice, this
 * list of conditions and the following disclaimer.
 *
 * Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimer in the documentation
 * and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 *ARE
 * DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR
 * SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER
 * CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY,
 * OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *****************************************************************************/

/******************************************************************************
 * Copyright 2018 The Apollo Authors. All Rights Reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *****************************************************************************/

#include <iostream>

#include "block_uf.h"
#include "texture.h"

namespace apollo {
namespace perception {
namespace block_uf {

__device__ int Find(int* parent, int i) {
  while (parent[i] != i) {
    i = parent[i];
  }
  return i;
}

__device__ void Union(int* parent, int i, int j) {
  bool done;
  do {
    i = Find(parent, i);
    j = Find(parent, j);

    if (i < j) {
      int old = atomicMin(&parent[j], i);
      done = (old == j);
      j = old;
    } else if (i > j) {
      int old = atomicMin(&parent[i], j);
      done = (old == i);
      i = old;
    } else {
      done = true;
    }
  } while (!done);
}

__global__ void BlockUnionFindInternal(int* label_array, int width,
                                       int height) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int global_index = y * width + x;
  int block_index = blockDim.x * threadIdx.y + threadIdx.x;

  __shared__ int s_parent[UF_BLOCK_WIDTH * UF_BLOCK_HEIGHT];
  __shared__ unsigned char s_img[UF_BLOCK_WIDTH * UF_BLOCK_HEIGHT];

  bool is_valid = x < width && y < height;

  s_parent[block_index] = block_index;
  s_img[block_index] = is_valid ? tex2D(img_tex, x, y) : 0xFF;
  __syncthreads();

  unsigned char v = s_img[block_index];

  if (is_valid && threadIdx.x > 0 && v != 0 && s_img[block_index - 1] == v) {
    Union(s_parent, block_index, block_index - 1);
  }
  __syncthreads();

  if (is_valid && threadIdx.y > 0 && v != 0 &&
      s_img[block_index - blockDim.x] == v) {
    Union(s_parent, block_index, block_index - blockDim.x);
  }
  __syncthreads();

  if (is_valid) {
    int f = Find(s_parent, block_index);
    int fx = f % UF_BLOCK_WIDTH;
    int fy = f / UF_BLOCK_WIDTH;
    label_array[global_index] =
        (blockIdx.y * blockDim.y + fy) * width +
            (blockIdx.x * blockDim.x + fx);
  }
}

__global__ void BlockUnionFindBoundary(int* label_array, int width,
                                       int height) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int global_index = y * width + x;

  bool is_valid = x < width && y < height;
  unsigned char v = is_valid ? tex2D(img_tex, x, y) : 0xFF;

  if (is_valid && y > 0 && threadIdx.y == 0 && v != 0 &&
      tex2D(img_tex, x, y - 1) == v) {
    Union(label_array, global_index, global_index - width);
  }

  if (is_valid && x > 0 && threadIdx.x == 0 && v != 0 &&
      tex2D(img_tex, x - 1, y) == v) {
    Union(label_array, global_index, global_index - 1);
  }
}

__global__ void BlockUnionFindRoot(int* label_array, int width, int height) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int global_index = y * width + x;

  bool is_valid = x < width && y < height;
  if (is_valid) {
    label_array[global_index] =
        tex2D(img_tex, x, y) > 0 ? Find(label_array, global_index) : -1;
  }
}

bool BlockUnionFind(const unsigned char* img, int width, int height,
                    int image_width, int* labels) {
  hipError_t cuda_err;

  hipChannelFormatDesc uchar_desc = hipCreateChannelDesc<unsigned char>();
  hipArray* img_array = NULL;
  hipMallocArray(&img_array, &uchar_desc, static_cast<size_t>(width),
                  static_cast<size_t>(height));
  hipBindTextureToArray(img_tex, img_array, uchar_desc);

  if (image_width == width) {
    size_t siz = static_cast<size_t>(width) * static_cast<size_t>(height) *
                 sizeof(unsigned char);
    hipMemcpyToArray(img_array, 0, 0, img, siz, hipMemcpyHostToDevice);
  } else {
    size_t siz = static_cast<size_t>(width) * sizeof(unsigned char);
    for (size_t i = 0; i < static_cast<size_t>(height); ++i) {
      hipMemcpyToArray(img_array, 0, i, img, siz, hipMemcpyHostToDevice);
      img += image_width;
    }
  }

  int* label_array;
  hipMalloc(
      (void**)&label_array,
      static_cast<size_t>(width) * static_cast<size_t>(height) * sizeof(int));

  dim3 block(UF_BLOCK_WIDTH, UF_BLOCK_HEIGHT);
  dim3 grid(
      static_cast<unsigned int>((width + UF_BLOCK_WIDTH - 1) / UF_BLOCK_WIDTH),
      static_cast<unsigned int>((height + UF_BLOCK_HEIGHT - 1) /
                                UF_BLOCK_HEIGHT));
  cuda_err = hipGetLastError();
  if (cuda_err != hipSuccess) {
    std::cerr << "failed to start block union find with CUDA: "
              << hipGetErrorString(cuda_err) << std::endl;
    return false;
  }

  hipDeviceSetCacheConfig(hipFuncCachePreferShared);

  BlockUnionFindInternal<<<grid, block>>>(label_array, width, height);

  hipDeviceSetCacheConfig(hipFuncCachePreferL1);

  BlockUnionFindBoundary<<<grid, block>>>(label_array, width, height);

  BlockUnionFindRoot<<<grid, block>>>(label_array, width, height);

  hipMemcpy(labels, label_array,
             static_cast<size_t>(width) *
                 static_cast<size_t>(height) * sizeof(int),
             hipMemcpyDeviceToHost);

  hipFree(label_array);
  hipFreeArray(img_array);
  cuda_err = hipGetLastError();
  if (cuda_err != hipSuccess) {
    std::cerr << "failed to end block union find with CUDA: "
              << hipGetErrorString(cuda_err) << std::endl;
    return false;
  }

  return true;
}

}  // namespace block_uf
}  // namespace perception
}  // namespace apollo
